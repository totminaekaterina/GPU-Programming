#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define SHARED_MEMORY


#define BLOCK_SIZE 16
#define BASE_TYPE double

#ifndef SHARED_MEMORY
__global__ void matrixMult(const BASE_TYPE *A, const BASE_TYPE *B, BASE_TYPE *C, int Acols, int Bcols)
{
    int i0 = Acols * (blockDim.y * blockIdx.y + threadIdx.y); // начальный индекс строки, которую должен обработать текущий поток
    int j0 = blockDim.x * blockIdx.x + threadIdx.x; 
    BASE_TYPE sum = 0; // сумма произведений
    for (int k = 0; k < Acols; k++)
        sum += A[i0 + k] * B[k * Bcols + j0];
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Индекс строки
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Индекс столбца
    int ind = row * Bcols + col; // Итоговый индекс в C

    C[ind] = sum;
}
#else
__global__ void matrixMult(const BASE_TYPE *A, const BASE_TYPE *B, BASE_TYPE *C, int Acols, int Bcols)
{
    // индекс начала первой подматрицы А, которую обрабатывает блок
    int aBegin = Acols * blockDim.y * blockIdx.y;
    // индекс конца подматрицы А, которую обрабатывает блок
    int aEnd = aBegin + Acols - 1;
    // шаг для перебора подматриц А
    int aStep = blockDim.x;
    // индекс начала первой подматрицы В, которую обрабатывает блок
    int bBegin = blockDim.x * blockIdx.x;
    // шаг для перебора подматриц В
    int bStep = blockDim.y * Bcols;

    // Выделение разделяемой памяти для подматриц
    __shared__ BASE_TYPE as[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ BASE_TYPE bs[BLOCK_SIZE][BLOCK_SIZE];
    
    // переменная для вычисления элемента подматрицы
    BASE_TYPE sum = 0.0;

    for (int ia = aBegin, ib = bBegin; ia < aEnd; ia += aStep, ib += bStep)
    {
        // загрузка подматриц А и В из глобальной памяти в разделяемую
        as[threadIdx.y][threadIdx.x] = A[ia + Acols * threadIdx.y + threadIdx.x];
        bs[threadIdx.y][threadIdx.x] = B[ib + Bcols * threadIdx.y + threadIdx.x];
        // Каждый поток загружает элемент из A и B в разделяемую память. 
        // Индексы потоков (threadIdx) и блоков (blockIdx) обеспечивают правильную адресацию
        
        // синхронизация нитей
        __syncthreads();
        
        // перемножение двух матриц
        for (int k = 0; k < blockDim.x; k++)
            sum += as[threadIdx.y][k] * bs[k][threadIdx.x];
        
        // синхронизация нитей
        __syncthreads();
    }
    
    // индекс результирующего элемента в глобальной памяти
    int ind = Bcols * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    
    // запись элемента в глобальную память
    C[ind] = sum;
}
#endif

int toMultiple(int a, int b)
{
    int mod = a % b;
    if (mod != 0)
    {
        mod = b - mod;
        return a + mod;
    }
    return a;
}



int main()
{
    #ifdef SHARED_MEMORY
        printf("Shared memory is enabled.\n");
    #else
        printf("Shared memory is not enabled.\n");
    #endif

    // start, stop - for Kernel time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // количество строк и столбцов матрицы
    int Arows = 1000;
    int Acols = 2000;
    int Brows = Acols;
    int Bcols = 1500;

    Arows = toMultiple(Arows, BLOCK_SIZE);
    printf("Arows = %d\n", Arows);

    Acols = toMultiple(Acols, BLOCK_SIZE);
    printf("Acols = %d\n", Acols);

    Brows = toMultiple(Brows, BLOCK_SIZE);
    printf("Brows = %d\n", Brows);

    Bcols = toMultiple(Bcols, BLOCK_SIZE);
    printf("Bcols = %d\n", Bcols);

    size_t Asize = Arows * Acols * sizeof(BASE_TYPE);
    size_t Bsize = Brows * Bcols * sizeof(BASE_TYPE);
    size_t Csize = Arows * Bcols * sizeof(BASE_TYPE);

    BASE_TYPE *h_A = (BASE_TYPE *)malloc(Asize);
    BASE_TYPE *h_B = (BASE_TYPE *)malloc(Bsize);
    BASE_TYPE *h_C = (BASE_TYPE *)malloc(Csize);

    for (int i = 0; i < Arows * Acols; ++i)
        h_A[i] = rand() / (BASE_TYPE)RAND_MAX;

    for (int i = 0; i < Brows * Bcols; ++i)
        h_B[i] = rand() / (BASE_TYPE)RAND_MAX;

    BASE_TYPE *d_A = NULL;
    hipMalloc((void **)&d_A, Asize);

    BASE_TYPE *d_B = NULL;
    hipMalloc((void **)&d_B, Bsize);

    BASE_TYPE *d_C = NULL;
    hipMalloc((void **)&d_C, Csize);

    hipMemcpy(d_A, h_A, Asize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Bsize, hipMemcpyHostToDevice);

    // параметры запуска ядра
    dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid = dim3(Bcols / BLOCK_SIZE, Arows / BLOCK_SIZE);

    hipEventRecord(start, 0);
    matrixMult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, Acols, Bcols);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float KernelTime;
    hipEventElapsedTime(&KernelTime, start, stop);

    float KernelTimeMicroseconds = KernelTime * 1000.0f;
    printf("KernelTime: %.2f microseconds\n", KernelTimeMicroseconds);

    hipMemcpy(h_C, d_C, Csize, hipMemcpyDeviceToHost);

    printf("Test STARTED\n");
    for (int i = 0; i < Arows; i++)
    {
        for (int j = 0; j < Bcols; j++)
        {
            BASE_TYPE sum = 0;
            for (int k = 0; k < Acols; k++)
                sum += h_A[i * Acols + k] * h_B[k * Bcols + j];
            if (fabs(h_C[i * Bcols + j] - sum) > 1e-3)
            {
                fprintf(stderr, "Result verification failed at element [%d, %d]!\n", i, j);
                printf("sum = %f, h_C[i * Bcols + j] = %f\n", sum, h_C[i * Bcols + j]);
                exit(EXIT_FAILURE);
            }
        }
    }
    printf("Test PASSED\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}